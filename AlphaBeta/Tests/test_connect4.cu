#include "hip/hip_runtime.h"
/*
	Test to check basic functioning of Connect-4 interface

	Rahul Kejriwal
	CS14B023
*/

#include <iostream>
#include "../GameInterfaces/Connect4.cu"
using namespace std;

__global__ void test_kernel(){
	GameState *initial_stage = new Connect4State;
	initial_stage = initial_stage->makeMove(0);
	initial_stage = initial_stage->makeMove(4);
	initial_stage = initial_stage->makeMove(4);
	initial_stage->printState();
}

void test_function(){
	GameState *initial_stage = new Connect4State;
	initial_stage = initial_stage->makeMove(0);
	initial_stage = initial_stage->makeMove(4);
	initial_stage = initial_stage->makeMove(4);
	initial_stage->printState();	
}

int main(){

	cout << "Checking Connect-4 interface" << endl 
		 << "----------------------------" << endl;

	cout << "Using Kernel: " << endl;
	test_kernel<<<1,1>>>();
	hipDeviceSynchronize();

	cout << "Using function: " << endl;
	test_function();

	cout << endl;

	return 0;
}