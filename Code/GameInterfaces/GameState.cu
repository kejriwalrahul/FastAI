
#include <hip/hip_runtime.h>
/*
	Generic Parent Class for all game interfaces

	Rahul Kejriwal
	CS14B023
*/

/*
	Abstract Class for abstracting actual game interface from game-playing algorithms
*/
class GameState {

protected:

	/*
		Array to hold moves from current GameState
		Can be used to generate children 
	*/
	bool *moves;
	int moves_length;

public:

	/*
		Evaluation function to be defined by concrete game interface
	*/
	__host__ __device__
	virtual int heuristicEval() = 0;

	/*
		Returns if the current game state is a terminal game tree node 
	*/
	__host__ __device__
	virtual bool isTerminal() = 0;

	/*
		Creates an array of possible moves in moves
	*/
	__host__ __device__
	virtual void moveGen() = 0;

	/*
		Returns the new game state after making the given move

		DANGER: No validity check for move # [Excersice Caution]
	*/
	__host__ __device__
	virtual GameState* makeMove(int) = 0;

	/*
		Prints Game Board for DEBUG purposes
	*/	
	__host__ __device__
	virtual void printState() = 0;
	
};