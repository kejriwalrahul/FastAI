#include <stdio.h>
#include "../GameInterfaces/TicTacToe.cu"
#include "../Includes/PriorityQueue.cu"
#include "../Includes/pq_kernels.cu"
#include <thrust/host_vector.h>

int main(){
	InsertTable *instab;
	DeleteTable *deltab;
	PriorityQueue *pq;
	
	hipHostAlloc((void **)&pq,sizeof(PriorityQueue),0);
	hipHostAlloc((void **)&instab,sizeof(InsertTable),0);
	hipHostAlloc((void **)&deltab,sizeof(DeleteTable),0);
	
	
	return 0;
}
