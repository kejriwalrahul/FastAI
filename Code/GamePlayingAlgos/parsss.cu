#include <stdio.h>
#include "../GameInterfaces/TicTacToe.cu"
#include "../Includes/PriorityQueue.cu"
#include "../Includes/pq_kernels.cu"
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#define BRANCH_FACTOR_TIC 10

int main(){
	InsertTable *h_instab,*d_instab;
	DeleteTable *h_deltab,*d_deltab;
	int h_offsets[QSIZE],*d_offsets;
	Node h_to_insert[NUM_PER_NODE*BRANCH_FACTOR_TIC];
	Node *d_to_insert;
	Node *d_to_send;
	int *num_inserts;
	PriorityQueue *h_pq,*d_pq;
	TicTacToeState *h_state,*d_state;
	
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMalloc((void **)&d_instab,sizeof(InsertTable));
	hipMalloc((void **)&d_deltab,sizeof(DeleteTable));
	hipMalloc((void **)&d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_TIC*sizeof(Node));
	hipMalloc((void **)&d_to_send,NUM_PER_NODE*sizeof(Node));
	hipMalloc((void **)&d_state,sizeof(TicTacToeState));
	hipMalloc(&d_offsets,QSIZE*sizeof(int));
	hipHostAlloc(&num_inserts,sizeof(int),0);
	
	h_instab = new InsertTable();
	h_deltab = new DeleteTable();
	h_pq = new PriorityQueue();
	
	Node node_list[2*NUM_PER_NODE];
	bool isInsertDone;
	int insertedSize;
	int num_indices;
	hipError_t err;int curr_size = 0;
	
	// Create root node
	/*Node root(INT_MAX-1,new TicTacToeState());
	node_list[0] = root;
	
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);*/
	
	//hipMemcpy(d_state,h_state,sizeof(TicTacToeState),hipMemcpyHostToDevice);
	h_offsets[0] = 0;
	h_offsets[1] = 3;
	//h_offsets[2] = 1;
	//h_offsets[3] = 4;
	hipMemcpy(d_offsets,h_offsets,2*sizeof(int), hipMemcpyHostToDevice);
	createRootNode<<<1,1>>>(d_to_insert,d_offsets,2);
	hipMemcpy(h_to_insert,d_to_insert,sizeof(Node), hipMemcpyDeviceToHost);
	insertedSize = 0;	
	do{
		
		h_instab->addEntry(0,h_to_insert+curr_size,1,h_pq->getInsertTarget(1,&isInsertDone,&insertedSize));
		curr_size += insertedSize;
	}while(!isInsertDone);
	hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
	hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
	hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	
	num_indices = 1;
	h_offsets[0] = 0;
	hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
	insert<<<1,1>>>(d_pq,d_instab,d_offsets,num_indices);
	hipDeviceSynchronize();
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	
	
	// At this stage the root node is present in the priority queue.
	
	bool *isEnd;
	hipHostAlloc(&isEnd,sizeof(bool),0);
	*isEnd = false;
	int time = 0;
	int num_to_process,num_to_insert,num_to_send;
	int target;
	PQNode curr_root;
	
	*num_inserts = 0;
	while(!(*isEnd)){
		hipMemcpy(h_to_insert,d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_TIC*sizeof(Node), hipMemcpyDeviceToHost);
		h_pq->print_object();
		hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		
		num_to_process = *num_inserts;
		curr_root = h_pq->readRoot();
		for(int i=0;i<curr_root.size;i++){
			h_to_insert[num_to_process++] = curr_root.nodes[i];
		}
		thrust::sort(h_to_insert,h_to_insert+num_to_process);
		
		num_to_insert = (num_to_process>NUM_PER_NODE)?num_to_process-NUM_PER_NODE:0;
		num_to_send = num_to_process-num_to_insert;
		//printf("%d %d %d Num processed\n",num_to_process,num_to_insert,num_to_send);
		h_pq->deleteUpdate(h_to_insert+num_to_send,num_to_insert,0);
		if(num_to_insert>0){
			h_deltab->addEntry();
		}
		num_to_insert -= NUM_PER_NODE;
		
		// Add the remaining to insert update.
		isInsertDone = false;
		curr_size = 0;
		insertedSize = 0;
		target = h_pq->getInsertTarget(num_to_insert,&isInsertDone,&insertedSize);
		while(num_to_insert>0){			
			h_instab->addEntry(0,h_to_insert+2*NUM_PER_NODE+curr_size,num_to_insert,target);
			curr_size += insertedSize;
			num_to_insert -= insertedSize;
			target++;
		}
		hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
		
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		// Delete update on even level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==0){
				h_offsets[num_indices++] = j;
			}
		}
		//printf("%d num deletes at even level\n",num_indices);
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
		delete_update<<<(num_indices+1023/1024),1024>>>(d_pq,d_deltab,d_offsets,num_indices);
		
		//hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		// Insert Update on even level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==0){
				h_offsets[num_indices++] = j;
			}
		}
		//printf("%d num inserts at even level\n",num_indices);
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
		insert<<<(num_indices+1023/1024),1024>>>(d_pq,d_instab,d_offsets,num_indices);
		
		//hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		//hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		// Delete update on odd level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==1){
				h_offsets[num_indices++] = j;
			}
		}
		//printf("%d num deletes at odd level\n",num_indices);
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
		delete_update<<<(num_indices+1023/1024),1024>>>(d_pq,d_deltab,d_offsets,num_indices);
		
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		// Insert Update on odd level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==1){
				h_offsets[num_indices++] = j;
			}
		}
		//printf("%d num inserts at odd level\n",num_indices);
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
		insert<<<(num_indices+1023/1024),1024>>>(d_pq,d_instab,d_offsets,num_indices);
		
		// Call the SSS* application, TODO - put it in a different stream
		hipMemcpy(d_to_send,h_to_insert,num_to_send*sizeof(Node),hipMemcpyHostToDevice);
		*num_inserts = 0;
		sss_star_algo<<<1,NUM_PER_NODE>>>(d_to_send,num_to_send,d_to_insert,num_inserts,isEnd);
		hipDeviceSynchronize();
		//printf("Num to insert: %d\n",*num_inserts);
		time++;
	}
	
	hipDeviceSynchronize();
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	//h_pq->print_object();
	//h_instab->printTable();
	//printf("%d\n",*isEnd);
	
	return 0;
}
