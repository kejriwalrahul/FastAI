#include <stdio.h>
#include "../GameInterfaces/TicTacToe.cu"
#include "../Includes/PriorityQueue.cu"
#include "../Includes/pq_kernels.cu"
#include <thrust/host_vector.h>

int main(){
	InsertTable *h_instab,*d_instab;
	DeleteTable *h_deltab,*d_deltab;
	int h_offsets[QSIZE],*d_offsets;
	PriorityQueue *h_pq,*d_pq;
	
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMalloc((void **)&d_instab,sizeof(InsertTable));
	hipMalloc((void **)&d_deltab,sizeof(DeleteTable));
	hipMalloc(&d_offsets,QSIZE*sizeof(int));
	
	h_instab = new InsertTable();
	h_deltab = new DeleteTable();
	h_pq = new PriorityQueue();
	
	Node node_list[2*NUM_PER_NODE];
	bool isInsertDone;
	int insertedSize;
	int num_indices;
	hipError_t err;
	
	// Create root node
	Node root(INT_MAX-1,new TicTacToeState());
	node_list[0] = root;
	insertedSize = 0;
	int curr_size = 0;
	do{
		
		h_instab->addEntry(0,node_list+curr_size,1,h_pq->getInsertTarget(1,&isInsertDone,&insertedSize));
		curr_size += insertedSize;
	}while(!isInsertDone);
	hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
	hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
	hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	
	num_indices = 1;
	h_offsets[0] = 0;
	hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
	insert<<<1,1>>>(d_pq,d_instab,d_offsets,num_indices);
	hipDeviceSynchronize();
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	
	
	// At this stage the root node is present in the priority queue.
	
	bool isEnd = false;
	int time = 0;
	PQNode curr_root;
	
	while(time<10){
		curr_root = h_pq->readRoot();
		
		time++;
	}
	
	
	
	
	return 0;
}
