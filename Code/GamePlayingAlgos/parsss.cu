#include <stdio.h>
#include "../GameInterfaces/TicTacToe.cu"
#include "../Includes/PriorityQueue.cu"
#include "../Includes/pq_kernels.cu"
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#define BRANCH_FACTOR_TIC 10

int main(){
	InsertTable *h_instab,*d_instab;
	DeleteTable *h_deltab,*d_deltab;
	int h_offsets[QSIZE],*d_offsets;
	Node h_to_insert[NUM_PER_NODE*BRANCH_FACTOR_TIC];
	Node *d_to_insert;
	int *num_inserts;
	PriorityQueue *h_pq,*d_pq;
	
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMalloc((void **)&d_instab,sizeof(InsertTable));
	hipMalloc((void **)&d_deltab,sizeof(DeleteTable));
	hipMalloc((void **)&d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_TIC*sizeof(Node));
	hipMalloc(&d_offsets,QSIZE*sizeof(int));
	hipHostAlloc(&num_inserts,sizeof(int),0);
	
	h_instab = new InsertTable();
	h_deltab = new DeleteTable();
	h_pq = new PriorityQueue();
	
	Node node_list[2*NUM_PER_NODE];
	bool isInsertDone;
	int insertedSize;
	int num_indices;
	hipError_t err;
	
	// Create root node
	Node root(INT_MAX-1,new TicTacToeState());
	node_list[0] = root;
	insertedSize = 0;
	int curr_size = 0;
	do{
		
		h_instab->addEntry(0,node_list+curr_size,1,h_pq->getInsertTarget(1,&isInsertDone,&insertedSize));
		curr_size += insertedSize;
	}while(!isInsertDone);
	hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
	hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
	hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	
	num_indices = 1;
	h_offsets[0] = 0;
	hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
	insert<<<1,1>>>(d_pq,d_instab,d_offsets,num_indices,num_inserts);
	hipDeviceSynchronize();
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	
	
	// At this stage the root node is present in the priority queue.
	
	bool isEnd = false;
	int time = 0;
	int num_to_process,num_to_insert,num_to_send;
	PQNode curr_root;
	
	*num_inserts = 0;
	while(time<10){
		hipMemcpy(h_to_insert,d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_TIC*sizeof(Node), hipMemcpyDeviceToHost);
		num_to_process = *num_inserts;
		curr_root = h_pq->readRoot();
		for(int i=0;i<curr_root.size;i++){
			h_to_insert[num_to_process++] = curr_root.nodes[i];
		}
		thrust::sort(h_to_insert,h_to_insert+num_to_process);
		num_to_insert = (num_to_process>NUM_PER_NODE)?num_to_process-NUM_PER_NODE:0;
		num_to_send = num_to_process-num_to_insert;
		h_pq->deleteUpdate(h_to_insert+num_to_send,num_to_insert,0);
		
		time++;
	}
	
	
	
	
	return 0;
}
