#include "PriorityQueue.h"
#include "pq_kernels.h"


int main(){
	PriorityQueue* pq = new PriorityQueue();
	PriorityQueue* d_pq;
	
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMemcpy(d_pq,pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	print<<<1,32>>>(d_pq);
	hipDeviceSynchronize();
	return 0;
}
