#include <stdio.h>
#include "../GameInterfaces/Connect4.cu"
#include "../Includes/PriorityQueue_C4.cu"
#include "../Includes/kernels_c4.cu"
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include "../Includes/timer.h"
#define BRANCH_FACTOR_C4 20
#define NUM_CANDIDATES 1000


int main(){
	InsertTable *h_instab,*d_instab;
	DeleteTable *h_deltab,*d_deltab;
	int h_offsets[QSIZE],*d_offsets;
	Node h_to_insert[NUM_PER_NODE*BRANCH_FACTOR_C4];
	Node *d_to_insert;
	Node *d_to_send;
	Node *d_candidates;
	int *num_inserts;
	int *bestMove;
	PriorityQueue *h_pq,*d_pq;
	Connect4State *d_state;
	char board[BOARD_SIZE];
	int col[NUM_COLS];
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMalloc((void **)&d_instab,sizeof(InsertTable));
	hipMalloc((void **)&d_deltab,sizeof(DeleteTable));
	hipMalloc((void **)&d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_C4*sizeof(Node));
	hipMalloc((void **)&d_to_send,NUM_PER_NODE*sizeof(Node));
	hipMalloc((void **)&d_candidates,NUM_CANDIDATES*sizeof(Node));
	hipMalloc((void **)&d_state,sizeof(Connect4State));
	hipMalloc(&d_offsets,QSIZE*sizeof(int));
	hipHostAlloc(&num_inserts,sizeof(int),0);
	hipHostAlloc(&bestMove,sizeof(int),0);
	
	h_instab = new InsertTable();
	h_deltab = new DeleteTable();
	h_pq = new PriorityQueue();
	CPUTimer cputimer;
	
	
	bool isInsertDone;
	int insertedSize;
	int num_indices;
	bool player;
	player = false;
	hipError_t err;
	int curr_size = 0;
	
	// Create root node
	
	//hipMemcpy(d_state,h_state,sizeof(Connect4State),hipMemcpyHostToDevice);
	int n,c,k;
	scanf("%d",&n);
	for(int i=0;i<BOARD_SIZE;i++){
		board[i] = '|';
	}
	for(int i=0;i<NUM_COLS;i++){
		col[i] = 0;
	}
	for(int i=0;i<n;i++){
		scanf("%d",&c);
		k = OFFSET(col[c],c);
		col[c]++;
		h_offsets[i] = k;
		if(i%2==0){
			board[k] = 'X';
		}
		else{
			board[k] = 'O';
		}
		player = !player;
	}
	printf("Initial Board\n");
	/*for(int i=0;i<NUM_ROWS;i++){
		for(int j=0;j<NUM_COLS;j++){
			printf("%c ",board[OFFSET(i,j)]);
		}
		printf("\n");
	}*/
	for(int i=NUM_ROWS-1; i>=0; i--){
		printf("|");
		for(int j=0; j<NUM_COLS; j++)
			printf("%c ", board[OFFSET(i,j)]);
		printf("\n");
	}
	
	cputimer.Start();
	hipMemcpy(d_offsets,h_offsets,n*sizeof(int), hipMemcpyHostToDevice);
	createRootNode<<<1,1>>>(d_to_insert,d_offsets,n);
	hipMemcpy(h_to_insert,d_to_insert,sizeof(Node), hipMemcpyDeviceToHost);
	insertedSize = 0;	
	do{
		
		h_instab->addEntry(0,h_to_insert+curr_size,1,h_pq->getInsertTarget(1,&isInsertDone,&insertedSize));
		curr_size += insertedSize;
	}while(!isInsertDone);
	hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
	hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
	hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	
	num_indices = 1;
	h_offsets[0] = 0;
	hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
	insert<<<1,1>>>(d_pq,d_instab,d_offsets,num_indices);
	hipDeviceSynchronize();
	hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	
	
	// At this stage the root node is present in the priority queue.
	
	
	// At this stage all candidates are in the priority queue.
	bool *isEnd;
	hipHostAlloc(&isEnd,sizeof(bool),0);
	*isEnd = false;
	
	*isEnd = false;
	int time = 0;
	int num_to_process,num_to_insert,num_to_send;
	int target;
	PQNode curr_root;
	
	*num_inserts = 0;
	int sum = 0;
	while(!(*isEnd)){
	//while(time < 20){
		hipMemcpy(h_to_insert,d_to_insert,NUM_PER_NODE*BRANCH_FACTOR_C4*sizeof(Node), hipMemcpyDeviceToHost);
		hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		
		num_to_process = *num_inserts;
		
		curr_root = h_pq->readRoot();
		for(int i=0;i<curr_root.size;i++){
			h_to_insert[num_to_process++] = curr_root.nodes[i];
		}
		thrust::stable_sort(h_to_insert,h_to_insert+num_to_process);
		
		num_to_send = (num_to_process>NUM_TO_SEND)?NUM_TO_SEND:num_to_process;
		num_to_insert = num_to_process - num_to_send;
		//num_to_send = num_to_process-num_to_insert;
		
		h_pq->deleteUpdate(h_to_insert+num_to_send,num_to_insert,0);
		if(num_to_insert>0){
			h_deltab->addEntry();
		}
		num_to_insert -= NUM_PER_NODE;
		
		
		// Add the remaining to insert update.
		isInsertDone = false;
		curr_size = 0;
		insertedSize = 0;
		target = h_pq->getInsertTarget(num_to_insert,&isInsertDone,&insertedSize);
		while(num_to_insert>0){			
			h_instab->addEntry(0,h_to_insert+NUM_TO_SEND+NUM_PER_NODE+curr_size,num_to_insert,target);
			curr_size += insertedSize;
			num_to_insert -= insertedSize;
			target++;
		}
		hipMemcpy(d_pq,h_pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
		
		//hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		// Delete update on even level
		num_indices = 0;
		//h_deltab->printTable();
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==0){
				h_offsets[num_indices++] = j;
			}
		}
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
		//printf("even deletes: %d\n",num_indices);
		delete_update<<<(num_indices+1023/1024),1024>>>(d_pq,d_deltab,d_offsets,num_indices);
		
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		// Insert Update on even level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==0){
				h_offsets[num_indices++] = j;
			}
		}
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
		insert<<<(num_indices+1023/1024),1024>>>(d_pq,d_instab,d_offsets,num_indices);
		//printf("even inserts: %d\n",num_indices);
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		// Delete update on odd level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==1){
				h_offsets[num_indices++] = j;
			}
		}
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
		delete_update<<<(num_indices+1023/1024),1024>>>(d_pq,d_deltab,d_offsets,num_indices);
		//printf("odd deletes: %d\n",num_indices);
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		// Insert Update on odd level
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==1){
				h_offsets[num_indices++] = j;
			}
		}
		hipMemcpy(d_offsets,h_offsets,num_indices*sizeof(int), hipMemcpyHostToDevice);
		//hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
		insert<<<(num_indices+1023/1024),1024>>>(d_pq,d_instab,d_offsets,num_indices);
		//printf("odd inserts: %d\n",num_indices);
		// Call the SSS* application, TODO - put it in a different stream
		hipMemcpy(d_to_send,h_to_insert,num_to_send*sizeof(Node),hipMemcpyHostToDevice);
		sum += num_to_send;
		*num_inserts = 0;
		sss_star_algo<<<1,NUM_PER_NODE>>>(d_to_send,num_to_send,d_to_insert,num_inserts,isEnd,bestMove,player);
		hipDeviceSynchronize();
		//err = hipGetLastError();
		//printf("error=%d, %s, %s\n", err, hipGetErrorName(err),hipGetErrorString(err));
		//printf("### %d %d\n",*num_inserts,curr_root.size);
		time++;
	}
	
	//hipDeviceSynchronize();
	//hipMemcpy(h_pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	cputimer.Stop();
	//printf("%d\n",*isEnd);
	//printf("%d\n",*bestMove);
	//printf("Iters: %d Nodes: %d\n",time,sum);
	if(n%2==0){
		board[OFFSET(col[*bestMove],*bestMove)] = 'X';
	}
	else{
		board[OFFSET(col[*bestMove],*bestMove)] = 'O';
	}
	printf("Final Board\n");
	for(int i=NUM_ROWS-1; i>=0; i--){
		printf("|");
		for(int j=0; j<NUM_COLS; j++)
			printf("%c ", board[OFFSET(i,j)]);
		printf("\n");
	}
	printf("Time taken: %lf milliseconds\n",cputimer.Elapsed()*1000);
	return 0;
}
