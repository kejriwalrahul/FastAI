#include <stdio.h>
#include "../GameInterfaces/GameState.cu"
#include "../Includes/PriorityQueue.cu"
#include "../Includes/pq_kernels.cu"
#include <stdlib.h>

int main(){
	PriorityQueue* pq = new PriorityQueue();
	PriorityQueue* d_pq;
	InsertTable* h_instab, *d_instab;
	DeleteTable* h_deltab, *d_deltab;
	h_instab = new InsertTable();
	h_deltab = new DeleteTable();
	/*InsertTable *instab;
	hipHostAlloc((void **)&instab,sizeof(InsertTable),0);
	instab = new InsertTable();*/
		
	hipMalloc((void **)&d_pq,sizeof(PriorityQueue));
	hipMemcpy(d_pq,pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_instab,sizeof(InsertTable));
	hipMalloc((void **)&d_deltab,sizeof(DeleteTable));
	hipError_t err;
	int vals[3];
	int indices[1000];
	int num_indices;
	int *d_indices;
	hipMalloc(&d_indices,1000*sizeof(int));
	for(int i=0;i<15;i++){
		//hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		for(int j=0;j<3;j++){
			vals[j] = rand()%100;
			printf("%d\n",vals[j]);
		}
		h_instab->addEntry(0,vals,3,i);
		//h_instab->printTable();
		hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
		
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==0){
				indices[num_indices++] = j;
			}
		}
		hipDeviceSynchronize();
		hipMemcpy(d_indices,indices,num_indices*sizeof(int),hipMemcpyHostToDevice);
		if(num_indices!=0)insert<<<1,num_indices>>>(d_pq,d_instab,d_indices,num_indices);
		
		hipDeviceSynchronize();
		/*err = hipGetLastError();
		printf("error=%d, %s, %s\n", err, hipGetErrorName(err),hipGetErrorString(err));*/
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_instab->status[j]==1 && h_instab->level[j]%2==1){
				indices[num_indices++] = j;
			}
		}
		if(num_indices!=0)insert<<<1,num_indices>>>(d_pq,d_instab,d_indices,num_indices);
		hipMemcpy(h_instab,d_instab,sizeof(InsertTable), hipMemcpyDeviceToHost);
		//h_instab->printTable();
		hipMemcpy(pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
		h_instab->printTable();
		pq->print_object();
		printf("########################\n");
	}
	/*int vals[3] = {3,4,5};
	h_instab->addEntry(0,vals,3,0);
	hipMemcpy(d_instab,h_instab,sizeof(InsertTable), hipMemcpyHostToDevice);
	print_val<<<1,4>>>(d_pq);
	int off[1] = {0};
	int *d_off;
	hipMalloc((void **)&d_off,sizeof(int));
	hipMemcpy(d_off,off,sizeof(int),hipMemcpyHostToDevice);
	writeToNode<<<1,4>>>(d_pq,d_instab,d_off,1);
	print_val<<<1,4>>>(d_pq);*/
	//print_val<<<1,1>>>(d_pq);
	hipDeviceSynchronize();
	//h_instab->printTable();
	hipMemcpy(pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
	pq->print_object();
	
	printf("HEllo %d\n",pq->curr_size);
	for(int j=0;j<3;j++){
			vals[j] = j*100;
			printf("%d\n",vals[j]);
		}
	vals[2] = INT_MAX;
	pq->deleteUpdate(vals,2,0);
	hipMemcpy(d_pq,pq,sizeof(PriorityQueue), hipMemcpyHostToDevice);
	h_deltab->addEntry();
	for(int i=0;i<2;i++){
		hipMemcpy(d_deltab,h_deltab,sizeof(DeleteTable), hipMemcpyHostToDevice);
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==0){
				indices[num_indices++] = j;
			}
		}
		hipMemcpy(d_indices,indices,num_indices*sizeof(int),hipMemcpyHostToDevice);
		if(num_indices!=0)delete_update<<<1,num_indices>>>(d_pq,d_deltab,d_indices,num_indices);
		hipMemcpy(pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
		pq->print_object();
		printf("########################\n");
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		num_indices = 0;
		for(int j=0;j<QSIZE;j++){
			if(h_deltab->status[j]==1 && h_deltab->level[j]%2==1){
				indices[num_indices++] = j;
			}
		}
		hipMemcpy(d_indices,indices,num_indices*sizeof(int),hipMemcpyHostToDevice);
		if(num_indices!=0)delete_update<<<1,num_indices>>>(d_pq,d_deltab,d_indices,num_indices);
		
		hipMemcpy(h_deltab,d_deltab,sizeof(DeleteTable), hipMemcpyDeviceToHost);
		hipMemcpy(pq,d_pq,sizeof(PriorityQueue), hipMemcpyDeviceToHost);
		pq->print_object();
		printf("########################\n");
	}
	
	return 0;
}
